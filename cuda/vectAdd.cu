
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vecAddKernel(float *A, float *B, float *C, int n){
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if(i<n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n){
  int size = n*sizeof(float);
  float *d_A, *d_B, *d_C;
  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  vecAddKernel<<<ceil(n/256.0), 256>>>(d_A,d_B,d_C,n);

  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

void datosRandom(float *array,int n){
  for(int i = 0; i < n; i++){
    array[i] = 1;
  }
}

void printVector(float *array,int n){
  for(int i = 0; i < n; i++){
    printf("%f ", array[i]);
  }
  printf("\n");
}

int main(){
  int n;
  float *h_A,*h_B,*h_C;
  scanf("%d", &n);
  h_A = (float*) malloc(n*sizeof(float));
  h_B = (float*) malloc(n*sizeof(float));
  h_C = (float*) malloc(n*sizeof(float));
  datosRandom(h_A,n);
  datosRandom(h_B,n);
  vecAdd(h_A,h_B,h_C,n);
  printVector(h_C,n);

  return 0;
}


