
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <time.h>

#define N 20
#define BLOCK_DIM 10
#define TILE_WIDTH 4

using namespace std;

// 4.2

__global__
void matMultKernel(int *d_M, int *d_N, int *d_P, int Width){
  int Row = blockIdx.y*blockDim.y + threadIdx.y;
  int Col = blockIdx.x*blockDim.x + threadIdx.x;
  int k = 0;
  if(Row < Width && Col < Width){
      float Pvalue = 0;
      for(k = 0; k < Width; ++k){
          Pvalue += d_M[Row*Width + k] * d_N[k*Width+Col];
      }
      d_P[Row*Width+Col] = Pvalue;
  }
}


// 4.5
__global__
void matMultKernel_tile(int *d_M, int *d_N, int *d_P, int Width){

  __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
 
  int Row = by*TILE_WIDTH + ty;
  int Col = bx*TILE_WIDTH + tx;

  float Pvalue = 0;
  int  ph,k;
  for(ph = 0; ph < Width/TILE_WIDTH; ++ph){
     Mds[ty][tx] = d_M[Row*Width+ph*TILE_WIDTH + tx];
     Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty) * Width + Col];
     __syncthreads();
     for(k = 0; k < TILE_WIDTH; ++k){
      Pvalue += Mds[ty][k] * Nds[k][tx];
     }
     __syncthreads();
  }
  d_P[Row*Width + Col] = Pvalue;
}

void imprimir_Matriz(int matrix[N][N]){
    for(int i=0;i<N;i++){
        for(int j=0; j<N; j++){
            cout<<matrix[i][j]<<' ';
        }
        cout<<endl;
    }
}

int main() {
    int a[N][N], b[N][N], c[N][N];
    int *dev_a, *dev_b, *dev_c;

    int size = N * N * sizeof(int);
    srand(time(NULL));
    for(int i=0; i<N; i++)
        for (int j=0; j<N; j++){
            a[i][j] = 1;
            b[i][j] = 1;
        }

    imprimir_Matriz(a);
    cout<<endl;
    imprimir_Matriz(b);


    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    
    
    dim3 dimGrid(ceil(N/4.0),ceil(N/4.0),1);
    dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
    //matMultKernel_tile<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c, N);     
    matMultKernel<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c, N);

    hipDeviceSynchronize();
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    
    cout<<endl;
    
    imprimir_Matriz(c);

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

    return 0;
}